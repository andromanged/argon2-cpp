// Copyright (c) 2019, Zpalmtree
//
// Please see the included LICENSE file for more information.


#include <hip/hip_runtime.h>
#include <cstring>
#include <stdint.h>
#include <iostream>

/* Sigma round constants */
__device__ __constant__ uint8_t SIGMA[12][16] = 
{
    { 0,  1,  2,  3,  4,  5,  6,  7,  8,  9,  10, 11, 12, 13, 14, 15 },
    { 14, 10, 4,  8,  9,  15, 13, 6,  1,  12, 0,  2,  11, 7,  5,  3  },
    { 11, 8,  12, 0,  5,  2,  15, 13, 10, 14, 3,  6,  7,  1,  9,  4  },
    { 7,  9,  3,  1,  13, 12, 11, 14, 2,  6,  5,  10, 4,  0,  15, 8  },
    { 9,  0,  5,  7,  2,  4,  10, 15, 14, 1,  11, 12, 6,  8,  3,  13 },
    { 2,  12, 6,  10, 0,  11, 8,  3,  4,  13, 7,  5,  15, 14, 1,  9  },
    { 12, 5,  1,  15, 14, 13, 4,  10, 0,  7,  6,  3,  9,  2,  8,  11 },
    { 13, 11, 7,  14, 12, 1,  3,  9,  5,  0,  15, 4,  8,  6,  2,  10 },
    { 6,  15, 14, 9,  11, 3,  0,  8,  12, 2,  13, 7,  1,  4,  10, 5  },
    { 10, 2,  8,  4,  7,  6,  1,  5,  15, 11, 9,  14, 3,  12, 13, 0  },
    { 0,  1,  2,  3,  4,  5,  6,  7,  8,  9,  10, 11, 12, 13, 14, 15 },
    { 14, 10, 4,  8,  9,  15, 13, 6,  1,  12, 0,  2,  11, 7,  5,  3  },
};

/* https://stackoverflow.com/a/13732181/8737306 */
template<typename T>
__device__ T rotateRight(T x, unsigned int moves)
{
    return (x >> moves) | (x << (sizeof(T) * 8 - moves));
}

__device__ __forceinline__
void mix(
    uint64_t &vA,
    uint64_t &vB,
    uint64_t &vC,
    uint64_t &vD,
    const uint64_t x,
    const uint64_t y)
{
    vA += vB + x;
    vD = rotateRight(vD ^ vA, 32);

    vC += vD;
    vB = rotateRight(vB ^ vC, 24);

    vA += vB + y;
    vD = rotateRight(vD ^ vA, 16);

    vC += vD;
    vB = rotateRight(vB ^ vC, 63);
}

__device__
void compress(uint64_t hash[8], uint64_t compressXorFlags[4], uint64_t chunk[16])
{
    /* Init hash with IV */
    uint64_t v[16] = 
    {
        hash[0],
        hash[1],
        hash[2],
        hash[3],
        hash[4],
        hash[5],
        hash[6],
        hash[7],
        0x6A09E667F3BCC908,
        0xBB67AE8584CAA73B,
        0x3C6EF372FE94F82B,
        0xA54FF53A5F1D36F1,
        0x510E527FADE682D1,
        0x9B05688C2B3E6C1F,
        0x1F83D9ABFB41BD6B,
        0x5BE0CD19137E2179,
    };

    v[12] ^= compressXorFlags[0];
    v[13] ^= compressXorFlags[1];
    v[14] ^= compressXorFlags[2];
    v[15] ^= compressXorFlags[3];

    for (int i = 0; i < 12; i++)
    {
        const auto &sigma = SIGMA[i];

        /* Column round */
        mix(v[0], v[4], v[8],  v[12], chunk[sigma[0]],  chunk[sigma[1]]);
        mix(v[1], v[5], v[9],  v[13], chunk[sigma[2]],  chunk[sigma[3]]);
        mix(v[2], v[6], v[10], v[14], chunk[sigma[4]],  chunk[sigma[5]]);
        mix(v[3], v[7], v[11], v[15], chunk[sigma[6]],  chunk[sigma[7]]);

        /* Diagonal round */
        mix(v[0], v[5], v[10], v[15], chunk[sigma[8]],  chunk[sigma[9]]);
        mix(v[1], v[6], v[11], v[12], chunk[sigma[10]], chunk[sigma[11]]);
        mix(v[2], v[7], v[8],  v[13], chunk[sigma[12]], chunk[sigma[13]]);
        mix(v[3], v[4], v[9],  v[14], chunk[sigma[14]], chunk[sigma[15]]);
    }

    for (int i = 0; i < 8; i++)
    {
        hash[i] ^= v[i] ^ v[i + 8];
    }
}

__device__
void blake2bGPU(
    uint8_t *result,
    uint8_t *input,
    size_t inputLength,
    uint8_t outputHashLength) /* Note: 1 to 64 bytes */
{
    uint64_t compressXorFlags[4] = {};

    /* Init hash with IV */
    uint64_t hash[8] = 
    {
        0x6A09E667F3BCC908,
        0xBB67AE8584CAA73B,
        0x3C6EF372FE94F82B,
        0xA54FF53A5F1D36F1,
        0x510E527FADE682D1,
        0x9B05688C2B3E6C1F,
        0x1F83D9ABFB41BD6B,
        0x5BE0CD19137E2179,
    };

    hash[0] ^= 0x01010000 ^ outputHashLength;

    uint64_t chunk[16] = {};

    uint8_t chunkSize = 0;

    size_t offset = 0;

    void *ptr = static_cast<void *>(&chunk[0]);

    while (inputLength > 0)
    {
        if (chunkSize == 0)
        {
            compress(hash, compressXorFlags, chunk);
            chunkSize = 0;
        }

        uint8_t size = 128 - chunkSize;

        if (size > inputLength)
        {
            size = static_cast<uint8_t>(inputLength);

            ptr = static_cast<uint8_t *>(ptr) + chunkSize;

            std::memcpy(ptr, input + offset, size);

            chunkSize += size;

            /* compressXorFlags[0..1] is a 128 bit number stored in little endian. */
            /* Increase the bottom bits */
            compressXorFlags[0] += size;

            /* If it's less than the value we just added, we overflowed, and need to
               add one to the top bits */
            compressXorFlags[1] += (compressXorFlags[0] < size) ? 1 : 0;

            inputLength -= size;

            offset += size;
        }
    }

    ptr = static_cast<void *>(&chunk[0]);
    ptr = static_cast<uint8_t *>(ptr) + chunkSize;

    /* Pad final chunk with zeros */
    std::memset(ptr, 0, 128 - chunkSize);

    /* Set all bytes, indicates last block */
    compressXorFlags[2] = 0xFFFFFFFFFFFFFFFF;

    /* Process final chunk */
    compress(hash, compressXorFlags, chunk);

    std::memcpy(result, &hash[0], outputHashLength);
}

void __global__
hashKernel(
    uint8_t *result,
    uint8_t *input,
    size_t inputLength,
    uint8_t outputHashLength) /* Note: 1 to 64 bytes */
{
    blake2bGPU(result, input, inputLength, outputHashLength);
}

/* input = 32 char byte array.
   output = 64 char hex string */
void byteArrayToHexString(const uint8_t *input, char *output)
{
    char hexval[16] = {'0', '1', '2', '3', '4', '5', '6', '7', '8', '9', 'a', 'b', 'c', 'd', 'e', 'f'};

    for (int i = 0; i < 64; i++)
    {
        output[i * 2] = hexval[((input[i] >> 4) & 0xF)];
        output[(i * 2) + 1] = hexval[(input[i]) & 0x0F];
    }
}

void hash()
{
    size_t outputHashLen = 64;

    uint8_t *result;

    hipMallocManaged(&result, outputHashLen * sizeof(uint8_t));

    std::cout << "Launching kernel" << std::endl;

    hashKernel<<<1, 1>>>(result, nullptr, 0, outputHashLen);

    hipDeviceSynchronize();
    
    char output[129];

    byteArrayToHexString(result, output);

    output[128] = '\0';

    std::cout << output << std::endl;

    hipFree(result);
}
