#include "hip/hip_runtime.h"
// Copyright (c) 2019, Zpalmtree
//
// Please see the included LICENSE file for more information.

#include <cstring>
#include <stdint.h>
#include <iostream>
#include <vector>

#include "Argon2.h"
#include "Blake2.h"

#define ERROR_CHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        std::string errorStr = hipGetErrorString(code);

        std::cout << "CUDA Error: " << errorStr << " at " << file << ", Line " << line << std::endl;

        if (abort)
        {
            throw std::runtime_error(errorStr);
        }
    }
}

struct block_th
{
    uint64_t a, b, c, d;
};

__device__ void move_block(struct block_th *dst, const struct block_th *src)
{
    *dst = *src;
}

__device__ void xor_block(struct block_th *dst, const struct block_th *src)
{
    dst->a ^= src->a;
    dst->b ^= src->b;
    dst->c ^= src->c;
    dst->d ^= src->d;
}

__device__ void load_block_cache(struct block_th *dst, const struct block_g *src, uint32_t thread)
{
    dst->a = src->data[0 * THREADS_PER_LANE + thread];
    dst->b = src->data[1 * THREADS_PER_LANE + thread];
    dst->c = src->data[2 * THREADS_PER_LANE + thread];
    dst->d = src->data[3 * THREADS_PER_LANE + thread];
}

__device__ void load_block_global(struct block_th *dst, const struct block_g *src, uint32_t thread)
{
    ulonglong2 *u128 = (ulonglong2*) src->data;
    asm("ld.global.ca.v2.u64 {%0, %1}, [%2];" : "=l"(dst->a), "=l"(dst->b) : "l"(&u128[0 * THREADS_PER_LANE + thread]));
    asm("ld.global.ca.v2.u64 {%0, %1}, [%2];" : "=l"(dst->c), "=l"(dst->d) : "l"(&u128[1 * THREADS_PER_LANE + thread]));
}

__device__ void store_block_cache(struct block_g *dst, const struct block_th *src, uint32_t thread)
{
    dst->data[0 * THREADS_PER_LANE + thread] = src->a;
    dst->data[1 * THREADS_PER_LANE + thread] = src->b;
    dst->data[2 * THREADS_PER_LANE + thread] = src->c;
    dst->data[3 * THREADS_PER_LANE + thread] = src->d;
}

__device__ void store_block_global(struct block_g *dst, const struct block_th *src, uint32_t thread)
{
    asm("st.global.wb.v2.u64 [%0], {%1, %2};" :: "l"(&dst->data[0 * THREADS_PER_LANE + 2 * thread]), "l"(src->a), "l"(src->b));
    asm("st.global.wb.v2.u64 [%0], {%1, %2};" :: "l"(&dst->data[2 * THREADS_PER_LANE + 2 * thread]), "l"(src->c), "l"(src->d));
}

__device__ void g(struct block_th *block)
{
    asm("{"
        ".reg .u64 s, x;"
        ".reg .u32 l1, l2, h1, h2;"
        // a = f(a, b);
        "add.u64 s, %0, %1;"            // s = a + b
        "cvt.u32.u64 l1, %0;"           // xlo = u64_lo(a)
        "cvt.u32.u64 l2, %1;"           // ylo = u64_lo(b)
        "mul.hi.u32 h1, l1, l2;"        // umulhi(xlo, ylo)
        "mul.lo.u32 l1, l1, l2;"        // xlo * ylo
        "mov.b64 x, {l1, h1};"          // x = u64_build(umulhi(xlo, ylo), xlo * ylo)
        "shl.b64 x, x, 1;"              // x = 2 * x
        "add.u64 %0, s, x;"             // a = s + x
        // d = rotr64(d ^ a, 32);
        "xor.b64 x, %3, %0;"
        "mov.b64 {h2, l2}, x;"
        "mov.b64 %3, {l2, h2};"         // swap hi and lo = rotr64(x, 32)
        // c = f(c, d);
        "add.u64 s, %2, %3;"
        "cvt.u32.u64 l1, %2;"
        "mul.hi.u32 h1, l1, l2;"
        "mul.lo.u32 l1, l1, l2;"
        "mov.b64 x, {l1, h1};"
        "shl.b64 x, x, 1;"
        "add.u64 %2, s, x;"
        // b = rotr64(b ^ c, 24);
        "xor.b64 x, %1, %2;"
        "mov.b64 {l1, h1}, x;"
        "prmt.b32 l2, l1, h1, 0x6543;"  // permute bytes 76543210 => 21076543
        "prmt.b32 h2, l1, h1, 0x2107;"  // rotr64(x, 24)
        "mov.b64 %1, {l2, h2};"
        // a = f(a, b);
        "add.u64 s, %0, %1;"
        "cvt.u32.u64 l1, %0;"
        "mul.hi.u32 h1, l1, l2;"
        "mul.lo.u32 l1, l1, l2;"
        "mov.b64 x, {l1, h1};"
        "shl.b64 x, x, 1;"
        "add.u64 %0, s, x;"
        // d = rotr64(d ^ a, 16);
        "xor.b64 x, %3, %0;"
        "mov.b64 {l1, h1}, x;"
        "prmt.b32 l2, l1, h1, 0x5432;"  // permute bytes 76543210 => 10765432
        "prmt.b32 h2, l1, h1, 0x1076;"  // rotr64(x, 16)
        "mov.b64 %3, {l2, h2};"
        // c = f(c, d);
        "add.u64 s, %2, %3;"
        "cvt.u32.u64 l1, %2;"
        "mul.hi.u32 h1, l1, l2;"
        "mul.lo.u32 l1, l1, l2;"
        "mov.b64 x, {l1, h1};"
        "shl.b64 x, x, 1;"
        "add.u64 %2, s, x;"
        // b = rotr64(b ^ c, 63);
        "xor.b64 x, %1, %2;"
        "shl.b64 s, x, 1;"              // x << 1
        "shr.b64 x, x, 63;"             // x >> 63
        "add.u64 %1, s, x;"             // emits less instructions than "or"
        "}"
        : "+l"(block->a), "+l"(block->b), "+l"(block->c), "+l"(block->d)
    );
}

__device__ void transpose1(struct block_th *block, uint32_t thread)
{
    uint32_t src_thr = (thread ^ 0x2);
    uint32_t g2 = (thread & 0x2);
    uint32_t g4 = (thread & 0x4);

    uint64_t xab = __shfl_sync(0xFFFFFFFF, g2 ? block->a : block->b, src_thr);
    uint64_t xcd = __shfl_sync(0xFFFFFFFF, g2 ? block->c : block->d, src_thr);

    uint64_t xa = g2 ? xab : block->a;
    uint64_t xc = g2 ? xcd : block->c;
    uint64_t xac = __shfl_xor_sync(0xFFFFFFFF, g4 ? xa : xc, 0x4);

    uint64_t xb = g2 ? block->b : xab;
    uint64_t xd = g2 ? block->d : xcd;
    uint64_t xbd = __shfl_xor_sync(0xFFFFFFFF, g4 ? xb : xd, 0x4);

    block->a = g4 ? xac : xa;
    block->b = g4 ? xbd : xb;
    block->c = g4 ? xc : xac;
    block->d = g4 ? xd : xbd;
}

__device__ void transpose2(struct block_th *block, uint32_t thread)
{
    uint32_t src_thr = (thread ^ 0x10);
    uint32_t g4 = (thread & 0x4);
    uint32_t g16 = (thread & 0x10);

    uint64_t xac = __shfl_xor_sync(0xFFFFFFFF, g4 ? block->a : block->c, 0x4);
    uint64_t xbd = __shfl_xor_sync(0xFFFFFFFF, g4 ? block->b : block->d, 0x4);

    uint64_t xa = g4 ? xac : block->a;
    uint64_t xb = g4 ? xbd : block->b;
    uint64_t xab = __shfl_sync(0xFFFFFFFF, g16 ? xa : xb, src_thr);

    uint64_t xc = g4 ? block->c : xac;
    uint64_t xd = g4 ? block->d : xbd;
    uint64_t xcd = __shfl_sync(0xFFFFFFFF, g16 ? xc : xd, src_thr);

    block->a = g16 ? xab : xa;
    block->b = g16 ? xb : xab;
    block->c = g16 ? xcd : xc;
    block->d = g16 ? xd : xcd;
}

__device__ void transpose3(struct block_th *block, uint32_t thread)
{
    uint32_t src_thr1 = (thread ^ 0x10);
    uint32_t src_thr2 = (thread ^ 0x2);
    uint32_t g2 = (thread & 0x2);
    uint32_t g16 = (thread & 0x10);

    uint64_t xab = __shfl_sync(0xFFFFFFFF, g16 ? block->a : block->b, src_thr1);
    uint64_t xcd = __shfl_sync(0xFFFFFFFF, g16 ? block->c : block->d, src_thr1);

    uint64_t xa = g16 ? xab : block->a;
    uint64_t xb = g16 ? block->b : xab;
    uint64_t xc = g16 ? xcd : block->c;
    uint64_t xd = g16 ? block->d : xcd;

    xab = __shfl_sync(0xFFFFFFFF, g2 ? xa : xb, src_thr2);
    xcd = __shfl_sync(0xFFFFFFFF, g2 ? xc : xd, src_thr2);

    block->a = g2 ? xab : xa;
    block->b = g2 ? xb : xab;
    block->c = g2 ? xcd : xc;
    block->d = g2 ? xd : xcd;
}

__device__ void shift1_shuffle(struct block_th *block, uint32_t thread)
{
    uint32_t mask = (thread & 0x2) >> 1;
    uint32_t src_thr_b = thread ^ mask ^ 0x2;
    uint32_t src_thr_d = thread ^ mask ^ 0x3;

    block->b = __shfl_sync(0xFFFFFFFF, block->b, src_thr_b, 0x4);
    block->c = __shfl_xor_sync(0xFFFFFFFF, block->c, 0x1, 0x4);
    block->d = __shfl_sync(0xFFFFFFFF, block->d, src_thr_d, 0x4);
}

__device__ void unshift1_shuffle(struct block_th *block, uint32_t thread)
{
    uint32_t mask = (thread & 0x2) >> 1;
    uint32_t src_thr_b = thread ^ mask ^ 0x3;
    uint32_t src_thr_d = thread ^ mask ^ 0x2;

    block->b = __shfl_sync(0xFFFFFFFF, block->b, src_thr_b, 0x4);
    block->c = __shfl_xor_sync(0xFFFFFFFF, block->c, 0x1, 0x4);
    block->d = __shfl_sync(0xFFFFFFFF, block->d, src_thr_d, 0x4);
}

__device__ void shift2_shuffle(struct block_th *block, uint32_t thread)
{
    uint32_t src_thr_b = thread ^ (((thread & 0x2) << 2) | 0x2);
    uint32_t src_thr_d = thread ^ (((~thread & 0x2) << 2) | 0x2);

    block->b = __shfl_sync(0xFFFFFFFF, block->b, src_thr_b);
    block->c = __shfl_xor_sync(0xFFFFFFFF, block->c, 0x8);
    block->d = __shfl_sync(0xFFFFFFFF, block->d, src_thr_d);
}

__device__ void unshift2_shuffle(struct block_th *block, uint32_t thread)
{
    uint32_t src_thr_b = thread ^ (((~thread & 0x2) << 2) | 0x2);
    uint32_t src_thr_d = thread ^ (((thread & 0x2) << 2) | 0x2);

    block->b = __shfl_sync(0xFFFFFFFF, block->b, src_thr_b);
    block->c = __shfl_xor_sync(0xFFFFFFFF, block->c, 0x8);
    block->d = __shfl_sync(0xFFFFFFFF, block->d, src_thr_d);
}

__device__ void shuffle_block(struct block_th *block, uint32_t thread)
{
    transpose1(block, thread);

    g(block);

    shift1_shuffle(block, thread);

    g(block);

    unshift1_shuffle(block, thread);
    transpose2(block, thread);

    g(block);

    shift2_shuffle(block, thread);

    g(block);

    unshift2_shuffle(block, thread);
    transpose3(block, thread);
}

__device__ uint32_t compute_ref_index(struct block_th *prev, uint32_t curr_index)
{
    uint32_t ref_index = __shfl_sync(0xFFFFFFFF, (uint32_t) prev->a, 0);

    uint32_t ref_area_size = curr_index - 1;
    ref_index = __umulhi(ref_index, ref_index);
    ref_index = ref_area_size - 1 - __umulhi(ref_area_size, ref_index);
    return ref_index;
}

__global__
void argon2Kernel(
    block_g *memory,
    uint32_t cache_size,
    uint32_t memory_tradeoff)
{
    extern __shared__ struct block_g cache[];

    // ref_index of the current block, -1 if current block is stored to global mem
    __shared__ uint16_t ref_indexes[TRTL_SCRATCHPAD_SIZE];

    uint32_t job_id = blockIdx.y;
    uint32_t thread = threadIdx.x;

    // select job's memory region
    memory += (size_t)job_id * TRTL_SCRATCHPAD_SIZE;

    struct block_th prev_prev, prev, ref, tmp;

    bool is_stored = true;

    load_block_global(&tmp, memory, thread);
    load_block_global(&prev, memory + 1, thread);

    // cache first block
    store_block_cache(&cache[0], &tmp, thread);
    uint32_t curr_cache_pos = 1;

    ((uint64_t*) ref_indexes)[0 * THREADS_PER_LANE + thread] = (uint64_t) -1;
    ((uint64_t*) ref_indexes)[1 * THREADS_PER_LANE + thread] = (uint64_t) -1;
    ((uint64_t*) ref_indexes)[2 * THREADS_PER_LANE + thread] = (uint64_t) -1;
    ((uint64_t*) ref_indexes)[3 * THREADS_PER_LANE + thread] = (uint64_t) -1;

    for (uint32_t curr_index = 2; curr_index < TRTL_SCRATCHPAD_SIZE; curr_index++)
    {
        move_block(&prev_prev, &prev);

        uint32_t ref_index = compute_ref_index(&prev, curr_index);
        uint32_t ref_ref_index = ref_indexes[ref_index];

        uint32_t ref_offset = curr_index - ref_index;

        if (ref_offset <= cache_size + 1)
        {
            uint32_t ref_cache_pos = curr_cache_pos + (cache_size + 1 - ref_offset);
            ref_cache_pos = (ref_cache_pos >= cache_size) ? ref_cache_pos - cache_size : ref_cache_pos;
            load_block_cache(&ref, &cache[ref_cache_pos], thread);
            xor_block(&prev, &ref);
        }
        else if (ref_ref_index == (uint16_t) -1)
        {
            load_block_global(&ref, memory + ref_index, thread);
            xor_block(&prev, &ref);
        }
        else
        {
            struct block_th ref_prev, ref_ref;

            load_block_global(&ref_prev, memory + ref_index - 1, thread);
            load_block_global(&ref_ref, memory + ref_ref_index, thread);
            xor_block(&ref_prev, &ref_ref);

            move_block(&tmp, &ref_prev);
            shuffle_block(&ref_prev, thread);
            xor_block(&ref_prev, &tmp);

            xor_block(&prev, &ref_prev);
        }

        move_block(&tmp, &prev);
        shuffle_block(&prev, thread);
        xor_block(&prev, &tmp);

        if (curr_index < TRTL_SCRATCHPAD_SIZE - 1)
        {
            if (curr_index > 2 + cache_size
                && ref_indexes[curr_index - cache_size - 1] == (uint16_t) -1)
            {
                load_block_cache(&tmp, &cache[curr_cache_pos], thread);
                store_block_global(memory + curr_index - cache_size - 1, &tmp, thread);
            }

            store_block_cache(&cache[curr_cache_pos], &prev_prev, thread);

            is_stored = !is_stored || (curr_index < memory_tradeoff) || (ref_ref_index != (uint16_t) -1);
            if (!is_stored)
            {
                ref_indexes[curr_index] = ref_index;
            }

            curr_cache_pos++;
            curr_cache_pos = (curr_cache_pos == cache_size) ? 0 : curr_cache_pos;
        }
    }

    store_block_global(memory + TRTL_SCRATCHPAD_SIZE - 1, &prev, thread);
}


kernelLaunchParams getLaunchParams(
    const uint32_t gpuIndex)
{
    kernelLaunchParams params;

    hipDeviceProp_t properties;

    /* Figure out how much memory we have available */
    hipGetDeviceProperties(&properties, gpuIndex);

    const size_t ONE_MB = 1024 * 1024;
    const size_t ONE_GB = ONE_MB * 1024;

    size_t memoryAvailable = (properties.totalGlobalMem / ONE_GB) * (ONE_GB / ONE_MB);

    /* The amount of nonces we're going to try per kernel launch */
    uint32_t noncesPerRun = (memoryAvailable * ONE_MB) / (sizeof(block_g) * TRTL_SCRATCHPAD_SIZE);
    noncesPerRun = (noncesPerRun / BLAKE_THREADS_PER_BLOCK) * BLAKE_THREADS_PER_BLOCK;

    /* The amount of memory we'll need to allocate on the GPU */
    params.memSize = sizeof(block_g) * TRTL_MEMORY * noncesPerRun;

    /* Init memory kernel params */
    params.initMemoryBlocks = noncesPerRun / BLAKE_THREADS_PER_BLOCK;
    params.initMemoryThreads = BLAKE_THREADS_PER_BLOCK;

    /* Argon2 kernel params */
    params.argon2Blocks = noncesPerRun;
    params.argon2Threads = THREADS_PER_LANE;
    params.argon2Cache = params.cache * ARGON_BLOCK_SIZE;

    params.getNonceBlocks = noncesPerRun / BLAKE_THREADS_PER_BLOCK;
    params.getNonceThreads = BLAKE_THREADS_PER_BLOCK;

    params.noncesPerRun = noncesPerRun;

    return params;
}

/**
 * Stuff we only need to do once (unless the algorithm changes).
 */
NvidiaState initializeState(const uint32_t gpuIndex)
{
    /* Set current device */
    ERROR_CHECK(hipSetDevice(gpuIndex));

    NvidiaState state;

    state.launchParams = getLaunchParams(gpuIndex);

    /* Allocate memory. These things will the be the same size for every job,
       unless the algorithm changes. */
    ERROR_CHECK(hipMalloc((void **)&state.memory, state.launchParams.memSize));
    ERROR_CHECK(hipMalloc((void **)&state.nonce, sizeof(uint32_t)));
    ERROR_CHECK(hipMalloc((void **)&state.hash, ARGON_HASH_LENGTH));
    ERROR_CHECK(hipMalloc((void **)&state.hashFound, sizeof(bool)));
    ERROR_CHECK(hipMalloc((void **)&state.blakeInput, BLAKE_BLOCK_SIZE * 2));

    ERROR_CHECK(hipMemset(state.hashFound, false, sizeof(bool)));
    ERROR_CHECK(hipMemset(state.nonce, 0, sizeof(uint32_t)));

    return state;
}

void freeState(NvidiaState &state)
{
    ERROR_CHECK(hipFree(state.memory));
    ERROR_CHECK(hipFree(state.nonce));
    ERROR_CHECK(hipFree(state.hash));
    ERROR_CHECK(hipFree(state.hashFound));
    ERROR_CHECK(hipFree(state.blakeInput));
}

void initJob(
    NvidiaState &state,
    const std::vector<uint8_t> &input,
    const std::vector<uint8_t> &saltInput,
    const uint32_t localNonce,
    const uint64_t target)
{
    state.localNonce = localNonce;
    state.target = target;

    setupBlakeInput(input, saltInput, state);
}

HashResult nvidiaHash(NvidiaState &state)
{
    /* Launch the first kernel to perform initial blake initialization */
    initMemoryKernel<<<
        dim3(state.launchParams.initMemoryBlocks),
        dim3(state.launchParams.initMemoryThreads, 2)
    >>>(
        state.memory,
        state.blakeInput,
        state.blakeInputSize,
        state.localNonce
    );

    /* Launch the second kernel to perform the main argon work */
    argon2Kernel<<<
        dim3(1, state.launchParams.argon2Blocks),
        dim3(state.launchParams.argon2Threads, 1),
        state.launchParams.argon2Cache
    >>>(
        state.memory,
        state.launchParams.cache,
        state.launchParams.memoryTradeoff
    );

    /* Launch the final kernel to perform final blake round and extract
       nonce that beats the target, if any */
    getNonceKernel<<<
        dim3(state.launchParams.getNonceBlocks),
        dim3(state.launchParams.getNonceThreads)
    >>>(
        state.memory,
        state.localNonce,
        state.target,
        state.nonce,
        state.hash,
        state.hashFound
    );

    /* Wait for kernel */
    ERROR_CHECK(hipPeekAtLastError());
    ERROR_CHECK(hipDeviceSynchronize());

    HashResult result;

    /* See if we found a valid nonce */
    ERROR_CHECK(hipMemcpy(&result.success, state.hashFound, sizeof(result.success), hipMemcpyDeviceToHost));
    
    if (result.success)
    {   
        /* Copy valid nonce + hash back to CPU */
        ERROR_CHECK(hipMemcpy(&result.nonce, state.nonce, sizeof(result.nonce), hipMemcpyDeviceToHost));
        ERROR_CHECK(hipMemcpy(&result.hash, state.hash, ARGON_HASH_LENGTH, hipMemcpyDeviceToHost));

        /* Clear the hash found flag so don't think we have found a share when we
           have not, along with the nonce */
        ERROR_CHECK(hipMemset(state.hashFound, false, sizeof(bool)));
        ERROR_CHECK(hipMemset(state.nonce, 0, sizeof(uint32_t)));
    }
    
    return result;
}
